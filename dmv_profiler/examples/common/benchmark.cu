
#include <iostream>
#include <cstdlib>
#include <fstream>

#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>

#include "ResultDatabase.h"
#include "OptionParser.h"
#include "Utility.h"
#include "cudacommon.h"
#include "libkineto.h"

using namespace std;

// Forward Declarations
void addBenchmarkSpecOptions(OptionParser &op);
void RunBenchmark(ResultDatabase &resultDB, OptionParser &op);

// ****************************************************************************
// Function: EnumerateDevicesAndChoose
//
// Purpose:
//   This function queries cuda about the available gpus in the system, prints
//   those results to standard out, and selects a device for use in the
//   benchmark.
//
// Arguments:
//   chooseDevice: logical number for the desired device
//   properties: whether or not to print device properties and exit
//
// Returns:  nothing
//
// Programmer: Jeremy Meredith
// Creation:
//
// Modifications:
//   Jeremy Meredith, Tue Oct  9 17:27:04 EDT 2012
//   Added a windows-specific --noprompt, which unless the user passes it,
//   prompts the user to press enter before the program exits on Windows.
//   This is because on Windows, the console disappears when the program
//   exits, but our results go to the console.
//
// ****************************************************************************
void EnumerateDevicesAndChoose(int chooseDevice, bool properties, bool quiet)
{
    hipSetDevice(chooseDevice);
    int actualdevice;
    hipGetDevice(&actualdevice);

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (properties)
    {
        cout << "Number of devices = " << deviceCount << "\n";
    }
    string deviceName = "";
    for (int device = 0; device < deviceCount; ++device)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        if (device == actualdevice)
            deviceName = deviceProp.name;
        if (properties)
        {
            cout << "Device " << device << ":\n";
            cout << "  name               = '" << deviceProp.name << "'"
                    << endl;
            cout << "  totalGlobalMem     = " << HumanReadable(
                    deviceProp.totalGlobalMem) << endl;
            cout << "  sharedMemPerBlock  = " << HumanReadable(
                    deviceProp.sharedMemPerBlock) << endl;
            cout << "  regsPerBlock       = " << deviceProp.regsPerBlock
                    << endl;
            cout << "  warpSize           = " << deviceProp.warpSize << endl;
            cout << "  memPitch           = " << HumanReadable(
                    deviceProp.memPitch) << endl;
            cout << "  maxThreadsPerBlock = " << deviceProp.maxThreadsPerBlock
                    << endl;
            cout << "  maxThreadsDim[3]   = " << deviceProp.maxThreadsDim[0]
                    << "," << deviceProp.maxThreadsDim[1] << ","
                    << deviceProp.maxThreadsDim[2] << endl;
            cout << "  maxGridSize[3]     = " << deviceProp.maxGridSize[0]
                    << "," << deviceProp.maxGridSize[1] << ","
                    << deviceProp.maxGridSize[2] << endl;
            cout << "  totalConstMem      = " << HumanReadable(
                    deviceProp.totalConstMem) << endl;
            cout << "  major (hw version) = " << deviceProp.major << endl;
            cout << "  minor (hw version) = " << deviceProp.minor << endl;
            cout << "  clockRate          = " << deviceProp.clockRate << endl;
            cout << "  textureAlignment   = " << deviceProp.textureAlignment
                    << endl;
        }
    }
    if(properties) {
        return;
    }
    if(!quiet) {
        cout << "Chose device:"
            << " name='"<<deviceName<<"'"
            << " index="<<actualdevice
            << endl;
    }
}

void checkCudaFeatureAvailability(OptionParser &op) {
    int device = 0;
    checkCudaErrors(hipGetDevice(&device));
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, device));
    
    // Check UVM availability
    if (op.getOptionBool("uvm") || op.getOptionBool("uvm-advise") ||
            op.getOptionBool("uvm-prefetch") || op.getOptionBool("uvm-prefetch-advise")) {
        if (!deviceProp.unifiedAddressing) {
            std::cerr << "device doesn't support unified addressing, exiting..." << std::endl;
            safe_exit(-1);
        }
    }

    // Check Cooperative Group availability
    if (op.getOptionBool("coop")) {
        if (!deviceProp.cooperativeLaunch) {
            std::cerr << "device doesn't support cooperative kernels, exiting..." << std::endl;
            safe_exit(-1);
        }
    }

    // Check Dynamic Parallelism availability
    if (op.getOptionBool("dyn")) {
        int runtimeVersion = 0;
        checkCudaErrors(hipRuntimeGetVersion(&runtimeVersion));
        if (runtimeVersion < 5000) {
            std::cerr << "CUDA runtime version less than 5.0, doesn't support \
                dynamic parallelism, exiting..." << std::endl;
            safe_exit(-1);
        }
    }

    // Check CUDA Graphs availability
    if (op.getOptionBool("graph")) {
        int runtimeVersion = 0;
        checkCudaErrors(hipRuntimeGetVersion(&runtimeVersion));
        if (runtimeVersion < 10000) {
            std::cerr << "CUDA runtime version less than 10.0, doesn't support \
                CUDA Graph, exiting..." << std::endl;
            safe_exit(-1);
        }
    }
}

// ****************************************************************************
// Function: main
//
// Purpose:
//   The main function takes care of initialization (device and MPI),  then
//   performs the benchmark and prints results.
//
// Arguments:
//
//
// Programmer: Jeremy Meredith
// Creation:
//
// Modifications:
//   Jeremy Meredith, Wed Nov 10 14:20:47 EST 2010
//   Split timing reports into detailed and summary.  For serial code, we
//   report all trial values, and for parallel, skip the per-process vals.
//   Also detect and print outliers from parallel runs.
//
// ****************************************************************************
int main(int argc, char *argv[])
{
    int ret = 0;

    try
    {
        // Get args
        OptionParser op;

        // Add shared options to the parser
        op.addOption("properties", OPT_BOOL, "0",
                "show properties for available platforms and devices", 'p');
        op.addOption("device", OPT_VECINT, "0",
                "specify device(s) to run on", 'd');
        op.addOption("passes", OPT_INT, "10", "specify number of passes", 'n');
        op.addOption("size", OPT_INT, "1", "specify problem size", 's');
        op.addOption("verbose", OPT_BOOL, "0", "enable verbose output", 'v');
        op.addOption("quiet", OPT_BOOL, "0", "enable concise output", 'q');
        op.addOption("configFile", OPT_STRING, "", "path of configuration file", 'c');
        op.addOption("inputFile", OPT_STRING, "", "path of input file", 'i');
        op.addOption("outputFile", OPT_STRING, "", "path of output file", 'o');
        op.addOption("metricsFile", OPT_STRING, "", "path of file to write metrics to", 'm');

        // Add options for turn on/off CUDA features
        op.addOption("uvm", OPT_BOOL, "0", "enable CUDA Unified Virtual Memory, only demand paging");
        op.addOption("uvm-advise", OPT_BOOL, "0", "guide the driver about memory usage patterns");
        op.addOption("uvm-prefetch", OPT_BOOL, "0", "prefetch memory the specified destination device");
        op.addOption("uvm-prefetch-advise", OPT_BOOL, "0", "prefetch memory the specified destination device with memory guidance on");
        op.addOption("coop", OPT_BOOL, "0", "enable CUDA Cooperative Groups");
        op.addOption("dyn", OPT_BOOL, "0", "enable CUDA Dynamic Parallelism");
        op.addOption("graph", OPT_BOOL, "0", "enable CUDA Graphs");
	op.addOption("traceFile", OPT_STRING, "./perf.json", "path to trace json", 't');

	// TODO: (surajk) move this to GEMM specific folder.
	op.addOption("field-length", OPT_INT, "128", "Field length for GEMM");
	op.addOption("kernel-version", OPT_INT, "0", "Kernel version for basic-gemm");
	op.addOption("fill-strategy", OPT_STRING, "default", "Filling strategy for GEMM");

        addBenchmarkSpecOptions(op);

        if (!op.parse(argc, argv))
        {
            op.usage();
            return (op.HelpRequested() ? 0 : 1);
        }

        bool properties = op.getOptionBool("properties");
        bool quiet = op.getOptionBool("quiet");
        string metricsfile = op.getOptionString("metricsFile");
	    string traceFile = op.getOptionString("traceFile");

        int device;
        device = op.getOptionVecInt("device")[0];
        int deviceCount;
        hipGetDeviceCount(&deviceCount);
        if (device >= deviceCount) {
            cerr << "Warning: device index: " << device <<
            " out of range, defaulting to device 0.\n";
            device = 0;
        }

        // Initialization
        EnumerateDevicesAndChoose(device, properties, quiet);
        if (properties)
        {
            return 0;
        }

        // Check CUDA feature availability
        checkCudaFeatureAvailability(op);

        ResultDatabase resultDB;

        // Add Profiler 
        // Kineto config
        std::set<libkineto::ActivityType> types = {
            libkineto::ActivityType::CONCURRENT_KERNEL,
            libkineto::ActivityType::GPU_MEMCPY,
            libkineto::ActivityType::GPU_MEMSET,
            libkineto::ActivityType::CUDA_RUNTIME,
            libkineto::ActivityType::EXTERNAL_CORRELATION,
        };

        std::vector<std::string> metrics = {
            "kineto__cuda_core_flops",
            "sm__inst_executed.sum",
            "l1tex__data_bank_conflicts_pipe_lsu.sum",
        };
        auto metricsConfigStr = fmt::format("CUPTI_PROFILER_METRICS = {}", fmt::join(metrics, ","));

        std::string profiler_config = "ACTIVITIES_WARMUP_PERIOD_SECS=0\n "
                                    metricsConfigStr
                                    "CUPTI_PROFILER_ENABLE_PER_KERNEL=true\n "
                                    "ACTIVITIES_DURATION_SECS=0";

        auto &profiler = libkineto::api().activityProfiler();
        libkineto::api().initProfilerIfRegistered();
        profiler.prepareTrace(types, profiler_config);
        auto isActive = profiler.isActive();

        profiler.startTrace();

        // Run the benchmark
        RunBenchmark(resultDB, op);

        auto trace = profiler.stopTrace();
        std::cout << "Stopped and processed trace. Got " << trace->activities()->size() << " activities.\n";
        trace->save(traceFile);

        // If quiet, output overall result
        // else output metrics
        if (quiet) {
            resultDB.DumpOverall();
        } else {

            if (metricsfile.empty()) {
                cout << endl;
                resultDB.DumpSummary(cout);
            } else {
                ofstream ofs;
                ofs.open(metricsfile.c_str(), std::ios_base::app);
                resultDB.DumpCsv(metricsfile);
                ofs.close();
            }
        }
    }
    catch( std::exception& e )
    {
        std::cerr << e.what() << std::endl;
        ret = 1;
    }
    catch( ... )
    {
        ret = 1;
    }

    return ret;
}
